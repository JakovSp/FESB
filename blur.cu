#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <chrono>
#include <ctime>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stdbool.h>
#include <math.h>

#include "include/types.h"
#include "include/stb_image.h"
#include "include/stb_image_write.h"
#include "include/gaussian.h"

using namespace std;
using namespace std::chrono;


float h_GaussianKernel[25] = {2, 4, 5, 4, 2,
							  4, 9, 12, 9, 4,
							  5, 12, 15, 12, 5,
							  4, 9, 12, 9, 4,
							  2, 4, 5, 4, 2};

#define KERNEL_RADIUS 2
#define TILE_SIZE (TILE_W + KERNEL_RADIUS * 2) * (TILE_W + KERNEL_RADIUS * 2)
#define KERNEL_SIZE 25
#define KERNEL_STRIDE 5

__device__ __constant__ float d_GaussianKernel[KERNEL_SIZE];

////////////////////////////////////////////////////////////

__global__ void Grayscale(Pixel3* dev_mem, uint8_t* out)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint8_t gray = (dev_mem[tid].red + dev_mem[tid].green + dev_mem[tid].blue) / 3;
	out[tid] = gray;
}

__global__ void convolutionGPU(uint8_t *d_Result, uint8_t *d_Data, int dataW, int dataH){

	// global mem address for this thread
	const int gLoc = threadIdx.x +
					 blockIdx.x * blockDim.x +
					 threadIdx.y * dataW +
					 blockIdx.y * blockDim.y * dataW;

	float sum = 0;
	float value = 0;

	for (int i = -KERNEL_RADIUS; i <= KERNEL_RADIUS; i++){
		for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++){
			
			if (blockIdx.x == 0 && (threadIdx.x + i) < 0)
				value = 0;
			else if ((blockIdx.x == (gridDim.x + 1)) && ((threadIdx.x + i) > blockDim.x + 1))
				value = 0;
			else
			{
				if ((blockIdx.y == 0) && ((threadIdx.y + j) < 0))
					value = 0;
				else if ((blockIdx.y == (gridDim.y + 1)) && ((threadIdx.y + j) > blockDim.y + 1))
					value = 0;
				else
					value = d_Data[gLoc + i + j * dataW];
			}
			sum += (value * d_GaussianKernel[KERNEL_RADIUS + i + KERNEL_STRIDE * (KERNEL_RADIUS + j)]);
		}
	}
	d_Result[gLoc] = (uint8_t)(sum / 159.0f);
}

void cpuGrayscale(Pixel3* in, uint8_t* out, int width, int height){

	int x, y;
	for (y = 0; y < height; ++y){
		for (x = 0; x < width; ++x){
			uint8_t gray = (in[x + y * width].red + in[x + y * width].green + in[x + y * width].blue) / 3;
			out[x + y * width] = gray;
		}
	}
}

void cpuConvolution(unsigned char* in, unsigned char* out, float* kernel, int nx, int ny, int radius){

	for (int m = radius; m < nx - radius; m++)
		for (int n = radius; n < ny - radius; n++){
			float pixel = 0.0;
			size_t c = 0;
			for (int j = -radius; j <= radius; j++)
				for (int i = -radius; i <= radius; i++){
					pixel += in[(n - j) * nx + m - i] * kernel[c];
					c++;
				}
			out[n * nx + m] = (unsigned char)(pixel / 159.0f);
		}
}


int main()
{

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int width, height, channels;
	Pixel3* h_Image = (Pixel3*)stbi_load("Slike/FESB1.jpg", &width, &height, &channels, STBI_rgb);
	unsigned char* GrayImage =	(unsigned char*)malloc(width * height);
	unsigned char* BlurImage =	(unsigned char*)malloc(width * height);

	auto startCPU = high_resolution_clock::now();

	cpuGrayscale(h_Image, GrayImage, width, height);
	cpuConvolution(GrayImage, BlurImage, h_GaussianKernel, width, height, KERNEL_RADIUS);

	auto stopCPU = high_resolution_clock::now();
	auto duration = duration_cast<milliseconds>(stopCPU - startCPU);
	cout << "CPU Execution Time: " << duration.count() << endl;

	stbi_write_jpg("blurCPU.jpg", width, height, 1, BlurImage, 100);

	///////////////////////////////////////////////////////////////////////////////////////////////////

	unsigned char* h_GPUBlur =	(unsigned char*)malloc(width * height);
	Pixel3* d_SourceImage;
	uint8_t* d_GrayChannel;
	uint8_t* d_BlurChannel;

	hipMalloc((void **)&d_SourceImage, width * height * channels * sizeof(uint8_t));
	hipMalloc((void **)&d_GrayChannel, width * height * sizeof(uint8_t));
	hipMalloc((void **)&d_BlurChannel, width * height * sizeof(uint8_t));
	hipMemcpy(d_SourceImage, h_Image, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemset(d_BlurChannel, 0, width * height);
	hipMemcpyToSymbol(HIP_SYMBOL(d_GaussianKernel), h_GaussianKernel, KERNEL_SIZE * sizeof(float));

	dim3 blocks(TILE_W, TILE_H);
	dim3 grids(width / TILE_W + 1, height / TILE_H + 1);

	hipEventRecord(start, 0);

	Grayscale<<<height, width>>>(d_SourceImage, d_GrayChannel);
	convolutionGPU<<<grids, blocks>>>(d_BlurChannel, d_GrayChannel, width, height);
	hipMemcpy(BlurImage, d_BlurChannel, width * height, hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("GPU Execution Time = %f\n", time);

 	stbi_write_jpg("blurGPU.jpg", width, height, 1, BlurImage, 100);

	///////////////////////////////////////////////////////////////////////////////////////////////////

	hipEventCreate(&start);
	hipEventCreate(&stop);
	uint8_t* d_BlurChannelFinal = d_GrayChannel;

	hipMemcpyToSymbol(HIP_SYMBOL(d_filtGauss), h_filtGauss, KERNEL_SIZE_SEP*sizeof(float));

	hipEventRecord(start, 0);
	convolutionRowGPU<<<grids, blocks>>>(d_BlurChannel, d_GrayChannel, width, height );
	// hipMemset(  d_BlurChannelFinal, 0, width * height);
	convolutionColGPU<<<grids, blocks>>>(d_BlurChannelFinal, d_BlurChannel, width, height );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("GPU Sep Execution Time = %f\n", time);

	hipMemcpy(BlurImage, d_BlurChannelFinal, width * height, hipMemcpyDeviceToHost );

 	stbi_write_jpg("blurGPUSep.jpg", width, height, 1, BlurImage, 100);

	///////////////////////////////////////////////////////////////////////////////////////////////////

	hipFree(d_SourceImage);
	hipFree(d_GrayChannel);
	hipFree(d_BlurChannel);

 	stbi_image_free(h_Image);
}
